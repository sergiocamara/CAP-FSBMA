#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
/*CUDA square matirx multiplication */
#define MIN(I, J) (I) < (J) ? (I) : (J)

uint64_t rtdsc() {
  uint32_t high, low;

  asm volatile("rdtsc" : "=a"(low), "=d"(high));
  return ((uint64_t)(high) << 32 | (low));
}

__global__ void multiply(int N, int *CC, int *AA, int *BB) {

#define TILE_WIDTH 32
#define A(I, J) AA[I * N + J]
#define B(I, J) BB[I * N + J]
#define C(I, J) CC[I * N + J]

  __shared__ int A_SharedMem[TILE_WIDTH][TILE_WIDTH];
  __shared__ int B_SharedMem[TILE_WIDTH][TILE_WIDTH];

  int accumulator = 0;
  int i, j;

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if(row <N && col < N){
    for (int k = 0; k < N; ++k)
	     C(row,col)+=A(row,k)*B(k,col);

  }
}

void initialize(int N, int *AA, int *BB) {
  int i, j;

#define A(I, J) AA[I * N + J]
#define B(I, J) BB[I * N + J]

  for (i = 0; i < N; ++i) {
    for (j = 0; j < N; ++j) {
      A(i, j) = random() % 4;
      B(i, j) = random() % 4;
    }
  }
}

int check(int N, int *CC, int *AA, int *BB) {
  int i, j, k;
  int sum;

#define A(I, J) AA[I * N + J]
#define B(I, J) BB[I * N + J]
#define C(I, J) CC[I * N + J]

  for (i = 0; i < N; ++i) {
    for (j = 0; j < N; ++j) {
      sum = 0;

      for (k = 0; k < N; ++k)
        sum = sum + A(i, k) * B(k, j);

      if (sum != C(i, j)){
	      printf("%d %d sum=%d C(i,j)=%d C(25,25)=%d \n",i,j,sum, C(i,j),C(25,25));
        return false;
      }

      }
    }

  return true;
}

int main(int argc, char *argv[]) {
  int N;
  int *A;
  int *B;
  int *C;

  int *A_DevPtr;
  int *B_DevPtr;
  int *C_DevPtr;

  uint64_t start, end;
  const char *result_str;

  srand(getpid());

  if (argc < 2) {
    fprintf(stderr, "USAGE: %s <N>", argv[0]);
    exit(1);
  }

  N = atoi(argv[1]);
  A = (int *)malloc(sizeof(int) * N * N);
  B = (int *)malloc(sizeof(int) * N * N);
  C = (int *)malloc(sizeof(int) * N * N);

if(  hipMalloc(&A_DevPtr, sizeof(int) * N * N)!=hipSuccess) {
	printf("Alloc Error A\n");return -1;
}
  if( hipMalloc(&B_DevPtr, sizeof(int) * N * N)!=hipSuccess){
	printf("Alloc Error B\n");return -1;
  hipFree(A_DevPtr);
	return -1;
  }
  if(hipMalloc(&C_DevPtr, sizeof(int) * N * N)!=hipSuccess){
	printf("Alloc Error C\n");
  hipFree(B_DevPtr);
  hipFree(A_DevPtr);
	return -1;
  }
printf("alloc done");
  initialize(N, A, B);

  if( hipMemcpy(A_DevPtr, A, sizeof(int) * N * N, hipMemcpyHostToDevice)!=hipSuccess){
	  printf("Failed copy of A");
  hipFree(C_DevPtr);
  hipFree(B_DevPtr);
  hipFree(A_DevPtr);
	return -1;
  }
  if(hipMemcpy(B_DevPtr, B, sizeof(int) * N * N, hipMemcpyHostToDevice)!=hipSuccess){
	  printf("Failed copy of B");
  hipFree(C_DevPtr);
  hipFree(B_DevPtr);
  hipFree(A_DevPtr);
	return -1;
  }
  if(hipMemset(C_DevPtr, 0, sizeof(int) * N * N)!=hipSuccess){
	  printf("Failed memset C");
  hipFree(C_DevPtr);
  hipFree(B_DevPtr);
  hipFree(A_DevPtr);
	return -1;
  }

  int NumberOfBlocksAxis = (N - 1) / 32 + 1;

  dim3 BlocksN(NumberOfBlocksAxis, NumberOfBlocksAxis, 1);
  dim3 ThreadsN(32, 32, 1);

  start = rtdsc();
  multiply<<<BlocksN, ThreadsN>>>(N, C_DevPtr, A_DevPtr, B_DevPtr);
hipDeviceSynchronize();
  end = rtdsc();
   hipError_t err = hipGetLastError();
   if(err!=hipSuccess){
        printf("CUDA Error: %s\n", hipGetErrorString(err));       

   }
  if(hipMemcpy(C, C_DevPtr, sizeof(int) * N * N, hipMemcpyDeviceToHost)!=hipSuccess){
	  printf("copy from GPU  error");
  }

  result_str = check(N, C, A, B) ? "Correct" : "Wrong";
  printf("Execution time: %ld cycles\n%s", end - start, result_str);

  hipFree(C_DevPtr);
  hipFree(B_DevPtr);
  hipFree(A_DevPtr);
  free(C);
  free(B);
  free(A);

  return 0;
}
